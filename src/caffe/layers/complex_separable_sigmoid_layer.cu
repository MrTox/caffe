#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/complex_separable_sigmoid_layer.hpp"

namespace caffe {

__device__ void sigmoid(const float x, float& result) {
   result = 1.0 / (1 + exp(-x) + 1e-14);
}

__device__ void sigmoid(const double x, double& result) {
   result = 1.0 / (1 + exp(-x) + 1e-14);
}

__global__ void ComplexSeparableSigmoidForward(const int n, const hipComplex* bottom, hipComplex* top) {
  CUDA_KERNEL_LOOP(index, n) {

    hipComplex z = bottom[index];

    float g_real = 0;
    float g_imag = 0;
    sigmoid(hipCrealf(z), g_real);
    sigmoid(hipCimagf(z), g_imag);

    top[index] = make_hipFloatComplex(g_real, g_imag);
  }
}

__global__ void ComplexSeparableSigmoidForward(const int n, const hipDoubleComplex* bottom, hipDoubleComplex* top) {
  CUDA_KERNEL_LOOP(index, n) {

    hipDoubleComplex z = bottom[index];

    double g_real = 0;
    double g_imag = 0;
    sigmoid(hipCreal(z), g_real);
    sigmoid(hipCimag(z), g_imag);

    top[index] = make_hipDoubleComplex(g_real, g_imag);
  }
}

__global__ void ComplexSeparableSigmoidBackward(const int n, const hipComplex* top,
    const hipComplex* top_diff, hipComplex* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
  
    // We already computed g_real = sigmoid(Re(bottom))
    // and g_imag = sigmoid(Imag(bottom)) during forward pass
    hipComplex g = top[index];
    float g_real = hipCrealf(g);
    float g_imag = hipCimagf(g);

    float dg_real = g_real * (1-g_real);
    float dg_imag = g_imag * (1-g_imag);

    hipComplex dfdz = make_hipFloatComplex(0.5*(dg_real + dg_imag), 0);
    hipComplex dfdcz = make_hipFloatComplex(0.5*(dg_real - dg_imag), 0);
    
    bottom_diff[index] = hipCaddf(
        hipCmulf(hipConjf(top_diff[index]), dfdcz),
        hipCmulf(top_diff[index], hipConjf(dfdz))
        );
  }
}

__global__ void ComplexSeparableSigmoidBackward(const int n, const hipDoubleComplex* top,
    const hipDoubleComplex* top_diff, hipDoubleComplex* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {

    // We already computed g_real = sigmoid(Re(bottom))
    // and g_imag = sigmoid(Imag(bottom)) during forward pass
    hipDoubleComplex g = top[index];
    double g_real = hipCreal(g);
    double g_imag = hipCimag(g);

    double dg_real = g_real * (1-g_real);
    double dg_imag = g_imag * (1-g_imag);

    hipDoubleComplex dfdz = make_hipDoubleComplex(0.5*(dg_real + dg_imag), 0);
    hipDoubleComplex dfdcz = make_hipDoubleComplex(0.5*(dg_real - dg_imag), 0);
 
    bottom_diff[index] = hipCadd(
        hipCmul(hipConj(top_diff[index]), dfdcz),
        hipCmul(top_diff[index], hipConj(dfdz))
        );
  }
}

template <>
void ComplexSeparableSigmoidLayer<float>::Forward_gpu(const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top) {
  const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<float>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexSeparableSigmoidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipComplex*)bottom_data, (hipComplex*)top_data);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexSeparableSigmoidLayer<double>::Forward_gpu(const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top) {
  const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<double>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexSeparableSigmoidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipDoubleComplex*)bottom_data, (hipDoubleComplex*)top_data);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexSeparableSigmoidLayer<float>::Backward_gpu(const vector<Blob<float>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<float>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<float>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<float>* top_data = this->RealToComplexTopData_gpu(top, 0);
    std::complex<float>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexSeparableSigmoidBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipComplex*)top_data, (const hipComplex*)top_diff, (hipComplex*)bottom_diff);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

template <>
void ComplexSeparableSigmoidLayer<double>::Backward_gpu(const vector<Blob<double>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<double>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<double>* top_data = this->RealToComplexTopData_gpu(top, 0);
    std::complex<double>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexSeparableSigmoidBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipDoubleComplex*)top_data, (const hipDoubleComplex*)top_diff, (hipDoubleComplex*)bottom_diff);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ComplexSeparableSigmoidLayer);

}  // namespace caffe
