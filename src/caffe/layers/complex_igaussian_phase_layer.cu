#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/complex_igaussian_phase_layer.hpp"

namespace caffe {

__global__ void ComplexIGaussianPhaseForward(const int n, const hipComplex* bottom, hipComplex* top, const float sigmaSq) {
  CUDA_KERNEL_LOOP(index, n) {
    hipComplex z = bottom[index];

    float zMag = hipCabsf(z) + 1e-14;
    float zReal = z.x;
    float zImag = z.y;
    float zSq = zReal*zReal + zImag*zImag;

    hipComplex g = make_hipFloatComplex(1 - expf(-zSq/(2*sigmaSq)), 0);
    hipComplex p = hipCdivf(z, make_hipFloatComplex(zMag,0) );
    top[index] = hipCmulf(g,p);
  }
}

__global__ void ComplexIGaussianPhaseForward(const int n, const hipDoubleComplex* bottom, hipDoubleComplex* top, const double sigmaSq) {
  CUDA_KERNEL_LOOP(index, n) {
    hipDoubleComplex z = bottom[index];

    double zMag = hipCabs(z) + 1e-14;
    double zReal = z.x;
    double zImag = z.y;
    double zSq = zReal*zReal + zImag*zImag;

    hipDoubleComplex g = make_hipDoubleComplex(1 - exp(-zSq/(2*sigmaSq)), 0);
    hipDoubleComplex p = hipCdiv(z, make_hipDoubleComplex(zMag,0) );
    top[index] = hipCmul(g,p);
  }
}

__global__ void ComplexIGaussianPhaseBackward(const int n, const hipComplex* bottom, const hipComplex* top_diff,
		hipComplex* bottom_diff, const float sigmaSq) {
  CUDA_KERNEL_LOOP(index, n) {

    hipComplex z = bottom[index];

    float zMag = hipCabsf(z) + 1e-14;
    float zReal = z.x;
    float zImag = z.y;
    float zSq = zReal*zReal + zImag*zImag;

    float g = 1 - expf(-zSq/(2*sigmaSq));
    hipComplex p = hipCdivf(z, make_hipFloatComplex(zMag,0) );

    hipComplex dgdcz = hipCmulf( make_hipFloatComplex((1-g)/(2*sigmaSq),0), z );
    hipComplex dgdz = hipConjf(dgdcz);

    hipComplex dpdz = make_hipFloatComplex(1/(2*zMag), 0);

    hipComplex conj_z = hipCaddf(hipConjf(z), make_hipFloatComplex(1e-14,0));
    hipComplex dpdcz = hipCmulf(make_hipFloatComplex(-0.5,0), hipCdivf(p , conj_z));

    hipComplex dfdz = hipCaddf( hipCmulf(p,dgdz), hipCmulf( make_hipFloatComplex(g,0), dpdz ) );
    hipComplex dfdcz = hipCaddf( hipCmulf(p,dgdcz), hipCmulf( make_hipFloatComplex(g,0), dpdcz ) );
    bottom_diff[index] = hipCaddf( hipCmulf(hipConjf(top_diff[index]),dfdcz), hipCmulf(top_diff[index],hipConjf(dfdz)) );
  }
}

__global__ void ComplexIGaussianPhaseBackward(const int n, const hipDoubleComplex* bottom, const hipDoubleComplex* top_diff,
		hipDoubleComplex* bottom_diff, const double sigmaSq) {
  CUDA_KERNEL_LOOP(index, n) {
    hipDoubleComplex z = bottom[index];

    double zMag = hipCabs(z) + 1e-14;
    double zReal = z.x;
    double zImag = z.y;
    double zSq = zReal*zReal + zImag*zImag;

    double g = 1 - exp(-zSq/(2*sigmaSq));
    hipDoubleComplex p = hipCdiv(z, make_hipDoubleComplex(zMag,0) );

    hipDoubleComplex dgdcz = hipCmul( make_hipDoubleComplex((1-g)/(2*sigmaSq),0), z );
    hipDoubleComplex dgdz = hipConj(dgdcz);

    hipDoubleComplex dpdz = make_hipDoubleComplex(1/(2*zMag), 0);

    hipDoubleComplex conj_z = hipCadd(hipConj(z), make_hipDoubleComplex(1e-14,0));
    hipDoubleComplex dpdcz = hipCmul(make_hipDoubleComplex(-0.5,0), hipCdiv(p , conj_z));

    hipDoubleComplex dfdz = hipCadd( hipCmul(p,dgdz), hipCmul( make_hipDoubleComplex(g,0), dpdz ) );
    hipDoubleComplex dfdcz = hipCadd( hipCmul(p,dgdcz), hipCmul( make_hipDoubleComplex(g,0), dpdcz ) );
    bottom_diff[index] = hipCadd( hipCmul(hipConj(top_diff[index]),dfdcz), hipCmul(top_diff[index],hipConj(dfdz)) );
  }
}

template <>
void ComplexIGaussianPhaseLayer<float>::Forward_gpu(const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top) {
  const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<float>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexIGaussianPhaseForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipComplex*)bottom_data, (hipComplex*)top_data, this->sigmaSq);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexIGaussianPhaseLayer<double>::Forward_gpu(const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top) {
  const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<double>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexIGaussianPhaseForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipDoubleComplex*)bottom_data, (hipDoubleComplex*)top_data, this->sigmaSq);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexIGaussianPhaseLayer<float>::Backward_gpu(const vector<Blob<float>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<float>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<float>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<float>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexIGaussianPhaseBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipComplex*)bottom_data, (const hipComplex*)top_diff, (hipComplex*)bottom_diff,
    		this->sigmaSq);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBottomDiff_gpu(bottom, 0);
  }
}

template <>
void ComplexIGaussianPhaseLayer<double>::Backward_gpu(const vector<Blob<double>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<double>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<double>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexIGaussianPhaseBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipDoubleComplex*)bottom_data, (const hipDoubleComplex*)top_diff, (hipDoubleComplex*)bottom_diff,
    		this->sigmaSq);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBottomDiff_gpu(bottom, 0);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ComplexIGaussianPhaseLayer);

}  // namespace caffe
