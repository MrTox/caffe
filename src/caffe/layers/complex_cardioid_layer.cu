#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/complex_cardioid_layer.hpp"

namespace caffe {

__global__ void ComplexCardioidForward(const int n, const hipComplex* bottom, hipComplex* top) {
  CUDA_KERNEL_LOOP(index, n) {
    hipComplex z = bottom[index];

    float theta = atan2f(z.y, z.x);
    top[index] = hipCmulf( make_hipFloatComplex(0.5f*(1+cosf(theta)),0), z );
  }
}

__global__ void ComplexCardioidForward(const int n, const hipDoubleComplex* bottom, hipDoubleComplex* top) {
  CUDA_KERNEL_LOOP(index, n) {
    hipDoubleComplex z = bottom[index];

    double theta = atan2(z.y, z.x);
    top[index] = hipCmul( make_hipDoubleComplex(0.5*(1+cos(theta)),0), z );
  }
}

__global__ void ComplexCardioidBackward(const int n, const hipComplex* bottom,
    const hipComplex* top_diff, hipComplex* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    hipComplex z = bottom[index];

    float theta = atan2f(z.y, z.x);

    hipComplex dfdz = make_hipFloatComplex(0.5f + 0.5f*cosf(theta), 0.25f*sinf(theta));

    hipComplex dfdcz = hipCdivf(
        hipCmulf(
            make_hipFloatComplex(0, -0.25f*sinf(theta)),
            z
            ),
        hipCaddf(hipConjf(z), make_hipFloatComplex(1e-14,0))
        );

    bottom_diff[index] = hipCaddf(
        hipCmulf(hipConjf(top_diff[index]), dfdcz),
        hipCmulf(top_diff[index], hipConjf(dfdz))
        );
  }
}

__global__ void ComplexCardioidBackward(const int n, const hipDoubleComplex* bottom,
    const hipDoubleComplex* top_diff, hipDoubleComplex* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    hipDoubleComplex z = bottom[index];

    double theta = atan2(z.y, z.x);

    hipDoubleComplex dfdz = make_hipDoubleComplex(0.5 + 0.5*cos(theta), 0.25*sin(theta));

    hipDoubleComplex dfdcz = hipCdiv(
        hipCmul(
            make_hipDoubleComplex(0, -0.25*sin(theta)),
            z
            ),
        hipCadd(hipConj(z), make_hipDoubleComplex(1e-14,0))
        );

    bottom_diff[index] = hipCadd(
        hipCmul(hipConj(top_diff[index]), dfdcz),
        hipCmul(top_diff[index], hipConj(dfdz))
        );
  }
}

template <>
void ComplexCardioidLayer<float>::Forward_gpu(const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top) {
  const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<float>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexCardioidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipComplex*)bottom_data, (hipComplex*)top_data);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexCardioidLayer<double>::Forward_gpu(const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top) {
  const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<double>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexCardioidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipDoubleComplex*)bottom_data, (hipDoubleComplex*)top_data);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexCardioidLayer<float>::Backward_gpu(const vector<Blob<float>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<float>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<float>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<float>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexCardioidBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipComplex*)bottom_data, (const hipComplex*)top_diff, (hipComplex*)bottom_diff);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

template <>
void ComplexCardioidLayer<double>::Backward_gpu(const vector<Blob<double>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<double>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<double>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexCardioidBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipDoubleComplex*)bottom_data, (const hipDoubleComplex*)top_diff, (hipDoubleComplex*)bottom_diff);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ComplexCardioidLayer);

}  // namespace caffe
