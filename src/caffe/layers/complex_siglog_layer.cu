#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/complex_siglog_layer.hpp"

namespace caffe {

__global__ void ComplexSiglogForward(const int n, const hipComplex* bottom, hipComplex* top,
    float d, float s, float r, float c) {
  CUDA_KERNEL_LOOP(index, n) {

    hipComplex z = bottom[index];

    hipComplex sz = hipCmulf(make_hipFloatComplex(s,0), z);
    float abs_sz = hipCabsf(sz);
    float sz_to_d = powf(abs_sz, d);
    float denom = c + 1.0/r * sz_to_d + 1e-14;
    top[index] = hipCmulf(sz, make_hipFloatComplex(1.0/denom, 0) );

  }
}

__global__ void ComplexSiglogForward(const int n, const hipDoubleComplex* bottom, hipDoubleComplex* top,
    double d, double s, double r, double c) {
  CUDA_KERNEL_LOOP(index, n) {

    hipDoubleComplex z = bottom[index];

    hipDoubleComplex sz = hipCmul(make_hipDoubleComplex(s,0), z);
    double abs_sz = hipCabs(sz);
    double sz_to_d = pow(abs_sz, d);
    double denom = c + 1.0/r * sz_to_d + 1e-14;
    top[index] = hipCmul(sz, make_hipDoubleComplex(1.0/denom, 0) );
  }
}

__global__ void ComplexSiglogBackward(const int n, const hipComplex* bottom,
    const hipComplex* top_diff, hipComplex* bottom_diff,
    float d, float s, float r, float c) {
  CUDA_KERNEL_LOOP(index, n) {

    hipComplex z = bottom[index];

    float abs_z = hipCabsf(z);

    hipComplex sz = hipCmulf(make_hipFloatComplex(s,0), z);
    float abs_sz = hipCabsf(sz);

    float z_to_d = powf(abs_z, d);
    float z_to_d_minus_one = powf(abs_z, d-1);
    float sz_to_d = powf(abs_sz, d);
    float s_to_d = powf(abs(s), d);

    // Useful temp variable
    float c_r_sz_d = c + 1.0/r * sz_to_d;

    float dfdz_numer = (c_r_sz_d)*s + s*(d/(2*r) * s_to_d * z_to_d);
    float dfdz_denom = c_r_sz_d * c_r_sz_d + 1e-14;
    hipComplex dfdz = make_hipFloatComplex(dfdz_numer/dfdz_denom, 0);

    float dfdcz_coeff_numer = -s * s_to_d * d * z_to_d_minus_one;
    float dfdcz_coeff_denom = 2 * abs_z * r * c_r_sz_d * c_r_sz_d + 1e-14;
    float dfdcz_coeff = dfdcz_coeff_numer / dfdcz_coeff_denom;
    hipComplex dfdcz = hipCmulf(make_hipFloatComplex(dfdcz_coeff, 0), hipCmulf(z,z));

    bottom_diff[index] = hipCaddf(
        hipCmulf(hipConjf(top_diff[index]), dfdcz),
        hipCmulf(top_diff[index], hipConjf(dfdz))
        );
  }
}

__global__ void ComplexSiglogBackward(const int n, const hipDoubleComplex* bottom,
    const hipDoubleComplex* top_diff, hipDoubleComplex* bottom_diff,
    double d, double s, double r, double c) {
  CUDA_KERNEL_LOOP(index, n) {

    hipDoubleComplex z = bottom[index];

    double abs_z = hipCabs(z);

    hipDoubleComplex sz = hipCmul(make_hipDoubleComplex(s,0), z);
    double abs_sz = hipCabs(sz);

    double z_to_d = pow(abs_z, d);
    double z_to_d_minus_one = pow(abs_z, d-1);
    double sz_to_d = pow(abs_sz, d);
    double s_to_d = pow(abs(s), d);

    // Useful temp variable
    double c_r_sz_d = c + 1.0/r * sz_to_d;

    double dfdz_numer = (c_r_sz_d)*s + s*(d/(2*r) * s_to_d * z_to_d);
    double dfdz_denom = c_r_sz_d * c_r_sz_d + 1e-14;
    hipDoubleComplex dfdz = make_hipDoubleComplex(dfdz_numer/dfdz_denom, 0);

    double dfdcz_coeff_numer = -s * s_to_d * d * z_to_d_minus_one;
    double dfdcz_coeff_denom = 2 * abs_z * r * c_r_sz_d * c_r_sz_d + 1e-14;
    double dfdcz_coeff = dfdcz_coeff_numer / dfdcz_coeff_denom;
    hipDoubleComplex dfdcz = hipCmul(make_hipDoubleComplex(dfdcz_coeff, 0), hipCmul(z,z));

    bottom_diff[index] = hipCadd(
        hipCmul(hipConj(top_diff[index]), dfdcz),
        hipCmul(top_diff[index], hipConj(dfdz))
        );
  }
}

template <>
void ComplexSiglogLayer<float>::Forward_gpu(const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top) {
  const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<float>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexSiglogForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipComplex*)bottom_data, (hipComplex*)top_data,
      this->d, this->s, this->r, this->c);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexSiglogLayer<double>::Forward_gpu(const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top) {
  const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
  std::complex<double>* top_data = this->RealToComplexTopData_mutable_gpu(top,0);

  int count = top[0]->count()/2;
  ComplexSiglogForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
		  (const hipDoubleComplex*)bottom_data, (hipDoubleComplex*)top_data,
      this->d, this->s, this->r, this->c);
  CUDA_POST_KERNEL_CHECK;

  this->SyncComplexTopData_gpu(top, 0);
}

template <>
void ComplexSiglogLayer<float>::Backward_gpu(const vector<Blob<float>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<float>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<float>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<float>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<float>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexSiglogBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipComplex*)bottom_data, (const hipComplex*)top_diff, (hipComplex*)bottom_diff,
        this->d, this->s, this->r, this->c);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

template <>
void ComplexSiglogLayer<double>::Backward_gpu(const vector<Blob<double>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom) {
  if (propagate_down[0]) {
    const std::complex<double>* top_diff = this->RealToComplexTopDiff_gpu(top,0);
    const std::complex<double>* bottom_data = this->RealToComplexBottomData_gpu(bottom, 0);
    std::complex<double>* bottom_diff = this->RealToComplexBottomDiff_mutable_gpu(bottom, 0);

    const int count = top[0]->count()/2;
    ComplexSiglogBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count,
    		(const hipDoubleComplex*)bottom_data, (const hipDoubleComplex*)top_diff, (hipDoubleComplex*)bottom_diff,
        this->d, this->s, this->r, this->c);
    CUDA_POST_KERNEL_CHECK;

    this->SyncComplexBlobDiff_gpu(0);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ComplexSiglogLayer);

}  // namespace caffe
