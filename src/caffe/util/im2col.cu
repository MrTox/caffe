#include "hip/hip_runtime.h"
#include <algorithm>
#include <complex>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        *data_col_ptr =
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
            data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <>
__global__ void im2col_gpu_kernel<hipComplex>(const int n, const hipComplex* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
	hipComplex* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;
    hipComplex* data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const hipComplex* data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i * dilation_h;
        int w_im = w_offset + j * dilation_w;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
        	*data_col_ptr = data_im_ptr[i * dilation_h * width + j * dilation_w];
        }
		else {
			data_col_ptr[0].x = 0;
			data_col_ptr[0].y = 0;
		}
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <>
__global__ void im2col_gpu_kernel<hipDoubleComplex>(const int n, const hipDoubleComplex* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w,
	const int stride_h, const int stride_w,
	const int dilation_h, const int dilation_w,
	const int height_col, const int width_col,
	hipDoubleComplex* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
	const int h_index = index / width_col;
	const int h_col = h_index % height_col;
	const int w_col = index % width_col;
	const int c_im = h_index / height_col;
	const int c_col = c_im * kernel_h * kernel_w;
	const int h_offset = h_col * stride_h - pad_h;
	const int w_offset = w_col * stride_w - pad_w;
	hipDoubleComplex* data_col_ptr = data_col;
	data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
	const hipDoubleComplex* data_im_ptr = data_im;
	data_im_ptr += (c_im * height + h_offset) * width + w_offset;
	for (int i = 0; i < kernel_h; ++i) {
	  for (int j = 0; j < kernel_w; ++j) {
		int h_im = h_offset + i * dilation_h;
		int w_im = w_offset + j * dilation_w;
		if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
		  *data_col_ptr = data_im_ptr[i * dilation_h * width + j * dilation_w];
		}
		else {
		  data_col_ptr[0].x = 0;
		  data_col_ptr[0].y = 0;
		}
		data_col_ptr += height_col * width_col;
	  }
	}
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h -
      (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w -
      (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w, double* data_col);

template <>
void im2col_gpu<std::complex<float> >(const std::complex<float>* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    std::complex<float>* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h -
      (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w -
      (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<hipComplex><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, (const hipComplex*)data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
      width_col, (hipComplex*)data_col);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void im2col_gpu<std::complex<double> >(const std::complex<double>* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    std::complex<double>* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h -
      (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
  int width_col = (width + 2 * pad_w -
      (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, (const hipDoubleComplex*)data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, dilation_h, dilation_w, height_col,
      width_col, (hipDoubleComplex*)data_col);
  CUDA_POST_KERNEL_CHECK;
}



template <typename Dtype, int num_axes>
__global__ void im2col_nd_gpu_kernel(const int n, const Dtype* data_im,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_col) {
  int d_temp[num_axes];  // NOLINT(runtime/arrays)
  int d_iter[num_axes];  // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  int i;
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % shared_col_shape[i + 1];
      channel_in /= shared_col_shape[i + 1];
      channel_out *= shared_kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= shared_col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * shared_stride[i] - shared_pad[i];
      channel_in *= shared_im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= shared_col_shape[i + 1];
      d_iter[i] = 0;
    }
    Dtype* data_col_ptr = data_col + channel_out;
    const Dtype* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] * shared_dilation[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < shared_im_shape[i + 1];
        if (!in_range) { break; }
      }
      if (in_range) {
        int data_im_offset = d_iter[0] * shared_dilation[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= shared_im_shape[i + 1];
          data_im_offset += d_iter[i] * shared_dilation[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        *data_col_ptr = 0;
      }
      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        const int d_max = shared_kernel_shape[i];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else {  // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);  // do
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <typename Dtype, int num_axes>
__global__ void im2col_nd_complex_gpu_kernel(const int n, const Dtype* data_im,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_col) {
  int d_temp[num_axes];  // NOLINT(runtime/arrays)
  int d_iter[num_axes];  // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  int i;
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % shared_col_shape[i + 1];
      channel_in /= shared_col_shape[i + 1];
      channel_out *= shared_kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= shared_col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * shared_stride[i] - shared_pad[i];
      channel_in *= shared_im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= shared_col_shape[i + 1];
      d_iter[i] = 0;
    }
    Dtype* data_col_ptr = data_col + channel_out;
    const Dtype* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] * shared_dilation[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < shared_im_shape[i + 1];
        if (!in_range) { break; }
      }
      if (in_range) {
        int data_im_offset = d_iter[0] * shared_dilation[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= shared_im_shape[i + 1];
          data_im_offset += d_iter[i] * shared_dilation[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        data_col_ptr[0].x = 0;
        data_col_ptr[0].y = 0;
      }
      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        const int d_max = shared_kernel_shape[i];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else {  // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);  // do
  }  // CUDA_KERNEL_LOOP(index, n)
}


template <typename Dtype>
void im2col_nd_gpu(const Dtype* data_im, const int num_spatial_axes,
    const int num_kernels, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_col) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    im2col_nd_gpu_kernel<Dtype, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 2:
    im2col_nd_gpu_kernel<Dtype, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 3:
    im2col_nd_gpu_kernel<Dtype, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 4:
    im2col_nd_gpu_kernel<Dtype, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 5:
    im2col_nd_gpu_kernel<Dtype, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 6:
    im2col_nd_gpu_kernel<Dtype, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 7:
    im2col_nd_gpu_kernel<Dtype, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 8:
    im2col_nd_gpu_kernel<Dtype, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 9:
    im2col_nd_gpu_kernel<Dtype, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  case 10:
    im2col_nd_gpu_kernel<Dtype, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, data_col);
    break;
  default:
    LOG(FATAL) << "im2col_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_nd_gpu<float>(const float* data_im,
    const int num_spatial_axes, const int col_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, float* data_col);
template void im2col_nd_gpu<double>(const double* data_im,
    const int num_spatial_axes, const int col_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, double* data_col);

template <>
void im2col_nd_gpu<std::complex<float> >(const std::complex<float>* data_im, const int num_spatial_axes,
    const int num_kernels, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, std::complex<float>* data_col) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    im2col_nd_complex_gpu_kernel<hipComplex, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 2:
    im2col_nd_complex_gpu_kernel<hipComplex, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 3:
    im2col_nd_complex_gpu_kernel<hipComplex, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 4:
    im2col_nd_complex_gpu_kernel<hipComplex, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 5:
    im2col_nd_complex_gpu_kernel<hipComplex, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 6:
    im2col_nd_complex_gpu_kernel<hipComplex, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 7:
    im2col_nd_complex_gpu_kernel<hipComplex, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 8:
    im2col_nd_complex_gpu_kernel<hipComplex, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 9:
    im2col_nd_complex_gpu_kernel<hipComplex, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  case 10:
    im2col_nd_complex_gpu_kernel<hipComplex, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipComplex*)data_col);
    break;
  default:
    LOG(FATAL) << "im2col_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

template <>
void im2col_nd_gpu<std::complex<double> >(const std::complex<double>* data_im, const int num_spatial_axes,
    const int num_kernels, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, std::complex<double>* data_col) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 2:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 3:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 4:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 5:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 6:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 7:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 8:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 9:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  case 10:
    im2col_nd_complex_gpu_kernel<hipDoubleComplex, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, (const hipDoubleComplex*)data_im, im_shape, col_shape,
        kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_col);
    break;
  default:
    LOG(FATAL) << "im2col_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                height_col + h_col) * width_col + w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

template <>
__global__ void col2im_gpu_kernel<hipComplex>(const int n, const hipComplex* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
	hipComplex* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
	hipComplex val = make_hipFloatComplex(0,0);
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                height_col + h_col) * width_col + w_col;
          val = hipCaddf(val, data_col[data_col_index]);
        }
      }
    }
    data_im[index] = val;
  }
}

template <>
__global__ void col2im_gpu_kernel<hipDoubleComplex>(const int n, const hipDoubleComplex* data_col,
    const int height, const int width, const int channels,
    const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int height_col, const int width_col,
	hipDoubleComplex* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
	hipDoubleComplex val = make_hipDoubleComplex(0,0);
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
                                height_col + h_col) * width_col + w_col;
          val = hipCadd(val, data_col[data_col_index]);
        }
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    Dtype* data_im) {
  int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
      stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
      stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
      pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    double* data_im);

template <>
void col2im_gpu<std::complex<float> >(const std::complex<float>* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, const int dilation_h, const int dilation_w,
    std::complex<float>* data_im) {
  int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
      stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
      stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<hipComplex><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, (const hipComplex*)data_col, height, width, channels, kernel_h, kernel_w,
      pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
      height_col, width_col, (hipComplex*)data_im);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void col2im_gpu<std::complex<double> >(const std::complex<double>* data_col, const int channels,
	const int height, const int width, const int kernel_h, const int kernel_w,
	const int pad_h, const int pad_w, const int stride_h,
	const int stride_w, const int dilation_h, const int dilation_w,
	std::complex<double>* data_im) {
  int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
	  stride_h + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
	  stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(num_kernels),
							 CAFFE_CUDA_NUM_THREADS>>>(
	  num_kernels, (const hipDoubleComplex*)data_col, height, width, channels, kernel_h, kernel_w,
	  pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
	  height_col, width_col, (hipDoubleComplex*)data_im);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype, int num_axes>
__global__ void col2im_nd_gpu_kernel(const int n, const Dtype* data_col,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_im) {
  int d_im[num_axes];  // NOLINT(runtime/arrays)
  int d_col_iter[num_axes];  // NOLINT(runtime/arrays)
  int d_col_start[num_axes];  // NOLINT(runtime/arrays)
  int d_col_end[num_axes];  // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int c_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = c_im % shared_im_shape[i + 1] + shared_pad[i];
      c_im /= shared_im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      const int kernel_extent =
          shared_dilation[i] * (shared_kernel_shape[i] - 1) + 1;
      d_col_start[i] = d_col_iter[i] =
          (d_im[i] < kernel_extent) ? 0 :
          (d_im[i] - kernel_extent) / shared_stride[i] + 1;
      d_col_end[i] =
          min(d_im[i] / shared_stride[i] + 1, shared_col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    Dtype val = 0;
    bool incremented = true;
    bool skip = false;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      int kernel_index;
      for (int i = num_axes - 1; i >= 0; --i) {
        kernel_index = d_im[i] - d_col_iter[i] * shared_stride[i];
        if (kernel_index % shared_dilation[i]) {
          skip = true;
          break;
        } else {
          kernel_index /= shared_dilation[i];
          final_offset += kernel_index * kernel_shape_prod;
          kernel_shape_prod *= shared_kernel_shape[i];
        }
      }
      if (!skip) {
        final_offset += kernel_shape_prod * c_im;
        for (int i = 0; i < num_axes; ++i) {
          final_offset *= shared_col_shape[i + 1];
          final_offset += d_col_iter[i];
        }
        val += data_col[final_offset];
      }
      skip = false;
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else {  // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    }  while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <int num_axes>
__global__ void col2im_nd_complex_gpu_kernel(const int n, const hipComplex* data_col,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, hipComplex* data_im) {
  int d_im[num_axes];  // NOLINT(runtime/arrays)
  int d_col_iter[num_axes];  // NOLINT(runtime/arrays)
  int d_col_start[num_axes];  // NOLINT(runtime/arrays)
  int d_col_end[num_axes];  // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int c_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = c_im % shared_im_shape[i + 1] + shared_pad[i];
      c_im /= shared_im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      const int kernel_extent =
          shared_dilation[i] * (shared_kernel_shape[i] - 1) + 1;
      d_col_start[i] = d_col_iter[i] =
          (d_im[i] < kernel_extent) ? 0 :
          (d_im[i] - kernel_extent) / shared_stride[i] + 1;
      d_col_end[i] =
          min(d_im[i] / shared_stride[i] + 1, shared_col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index].x = 0;
        data_im[index].y = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    hipComplex val = make_hipFloatComplex(0,0);
    bool incremented = true;
    bool skip = false;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      int kernel_index;
      for (int i = num_axes - 1; i >= 0; --i) {
        kernel_index = d_im[i] - d_col_iter[i] * shared_stride[i];
        if (kernel_index % shared_dilation[i]) {
          skip = true;
          break;
        } else {
          kernel_index /= shared_dilation[i];
          final_offset += kernel_index * kernel_shape_prod;
          kernel_shape_prod *= shared_kernel_shape[i];
        }
      }
      if (!skip) {
        final_offset += kernel_shape_prod * c_im;
        for (int i = 0; i < num_axes; ++i) {
          final_offset *= shared_col_shape[i + 1];
          final_offset += d_col_iter[i];
        }
        val = hipCaddf(val, data_col[final_offset]);
      }
      skip = false;
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else {  // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    }  while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <int num_axes>
__global__ void col2im_nd_complex_gpu_kernel(const int n, const hipDoubleComplex* data_col,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, hipDoubleComplex* data_im) {
  int d_im[num_axes];  // NOLINT(runtime/arrays)
  int d_col_iter[num_axes];  // NOLINT(runtime/arrays)
  int d_col_start[num_axes];  // NOLINT(runtime/arrays)
  int d_col_end[num_axes];  // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int c_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = c_im % shared_im_shape[i + 1] + shared_pad[i];
      c_im /= shared_im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      const int kernel_extent =
          shared_dilation[i] * (shared_kernel_shape[i] - 1) + 1;
      d_col_start[i] = d_col_iter[i] =
          (d_im[i] < kernel_extent) ? 0 :
          (d_im[i] - kernel_extent) / shared_stride[i] + 1;
      d_col_end[i] =
          min(d_im[i] / shared_stride[i] + 1, shared_col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index].x = 0;
        data_im[index].y = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    hipDoubleComplex val = make_hipDoubleComplex(0,0);
    bool incremented = true;
    bool skip = false;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      int kernel_index;
      for (int i = num_axes - 1; i >= 0; --i) {
        kernel_index = d_im[i] - d_col_iter[i] * shared_stride[i];
        if (kernel_index % shared_dilation[i]) {
          skip = true;
          break;
        } else {
          kernel_index /= shared_dilation[i];
          final_offset += kernel_index * kernel_shape_prod;
          kernel_shape_prod *= shared_kernel_shape[i];
        }
      }
      if (!skip) {
        final_offset += kernel_shape_prod * c_im;
        for (int i = 0; i < num_axes; ++i) {
          final_offset *= shared_col_shape[i + 1];
          final_offset += d_col_iter[i];
        }
        val = hipCadd(val, data_col[final_offset]);
      }
      skip = false;
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else {  // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    }  while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <typename Dtype>
void col2im_nd_gpu(const Dtype* data_col, const int num_spatial_axes,
    const int im_size, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, Dtype* data_im) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    col2im_nd_gpu_kernel<Dtype, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 2:
    col2im_nd_gpu_kernel<Dtype, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 3:
    col2im_nd_gpu_kernel<Dtype, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 4:
    col2im_nd_gpu_kernel<Dtype, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 5:
    col2im_nd_gpu_kernel<Dtype, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 6:
    col2im_nd_gpu_kernel<Dtype, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 7:
    col2im_nd_gpu_kernel<Dtype, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 8:
    col2im_nd_gpu_kernel<Dtype, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 9:
    col2im_nd_gpu_kernel<Dtype, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  case 10:
    col2im_nd_gpu_kernel<Dtype, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, data_im);
    break;
  default:
    LOG(FATAL) << "col2im_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_nd_gpu<float>(const float* data_col,
    const int num_spatial_axes, const int im_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, float* data_im);
template void col2im_nd_gpu<double>(const double* data_col,
    const int num_spatial_axes, const int im_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, double* data_im);

template <>
void col2im_nd_gpu<std::complex<float> >(const std::complex<float>* data_col, const int num_spatial_axes,
    const int im_size, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, std::complex<float>* data_im) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    col2im_nd_complex_gpu_kernel<1>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 2:
    col2im_nd_complex_gpu_kernel<2>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 3:
    col2im_nd_complex_gpu_kernel<3>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 4:
    col2im_nd_complex_gpu_kernel<4>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 5:
    col2im_nd_complex_gpu_kernel<5>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 6:
    col2im_nd_complex_gpu_kernel<6>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 7:
    col2im_nd_complex_gpu_kernel<7>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 8:
    col2im_nd_complex_gpu_kernel<8>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 9:
    col2im_nd_complex_gpu_kernel<9>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  case 10:
    col2im_nd_complex_gpu_kernel<10>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipComplex*)data_im);
    break;
  default:
    LOG(FATAL) << "col2im_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

template <>
void col2im_nd_gpu<std::complex<double> >(const std::complex<double>* data_col, const int num_spatial_axes,
    const int im_size, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    const int* dilation, std::complex<double>* data_im) {
  // num_axes should be smaller than block size
  DCHECK_LT(num_spatial_axes, CAFFE_CUDA_NUM_THREADS);
  switch (num_spatial_axes) {
  case 1:
    col2im_nd_complex_gpu_kernel<1>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 2:
    col2im_nd_complex_gpu_kernel<2>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 3:
    col2im_nd_complex_gpu_kernel<3>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 4:
    col2im_nd_complex_gpu_kernel<4>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 5:
    col2im_nd_complex_gpu_kernel<5>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 6:
    col2im_nd_complex_gpu_kernel<6>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 7:
    col2im_nd_complex_gpu_kernel<7>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 8:
    col2im_nd_complex_gpu_kernel<8>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 9:
    col2im_nd_complex_gpu_kernel<9>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  case 10:
    col2im_nd_complex_gpu_kernel<10>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, (const hipDoubleComplex*)data_col, im_shape, col_shape,
          kernel_shape, pad, stride, dilation, (hipDoubleComplex*)data_im);
    break;
  default:
    LOG(FATAL) << "col2im_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

}  // namespace caffe
