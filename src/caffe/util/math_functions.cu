#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<std::complex<float> >(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const std::complex<float> alpha, const std::complex<float>* A, const std::complex<float>* B,
    const std::complex<float> beta, std::complex<float>* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA;
  if(TransA == CblasNoTrans) {
    cuTransA = HIPBLAS_OP_N;
  }
  else if(TransA == CblasTrans){
    cuTransA = HIPBLAS_OP_T;
  }
  else {
    cuTransA = HIPBLAS_OP_C;
  }
  hipblasOperation_t cuTransB;
  if(TransB == CblasNoTrans) {
    cuTransB = HIPBLAS_OP_N;
  }
  else if(TransB == CblasTrans){
    cuTransB = HIPBLAS_OP_T;
  }
  else {
    cuTransB = HIPBLAS_OP_C;
  }
  CUBLAS_CHECK(hipblasCgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, (const hipComplex*)&alpha, (const hipComplex*)B, ldb, (const hipComplex*)A, lda,
      (const hipComplex*)&beta, (hipComplex*)C, N));
}

template <>
void caffe_gpu_gemm<std::complex<double> >(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const std::complex<double> alpha, const std::complex<double>* A, const std::complex<double>* B,
    const std::complex<double> beta, std::complex<double>* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA;
  if(TransA == CblasNoTrans) {
    cuTransA = HIPBLAS_OP_N;
  }
  else if(TransA == CblasTrans){
    cuTransA = HIPBLAS_OP_T;
  }
  else {
    cuTransA = HIPBLAS_OP_C;
  }
  hipblasOperation_t cuTransB;
  if(TransB == CblasNoTrans) {
    cuTransB = HIPBLAS_OP_N;
  }
  else if(TransB == CblasTrans){
    cuTransB = HIPBLAS_OP_T;
  }
  else {
    cuTransB = HIPBLAS_OP_C;
  }
  CUBLAS_CHECK(hipblasZgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, (const hipDoubleComplex*)&alpha, (const hipDoubleComplex*)B, ldb, (const hipDoubleComplex*)A, lda,
      (const hipDoubleComplex*)&beta, (hipDoubleComplex*)C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<std::complex<float> >(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const std::complex<float> alpha, const std::complex<float>* A, const std::complex<float>* x,
    const std::complex<float> beta, std::complex<float>* y) {
  hipblasOperation_t cuTransA;
  if(TransA == CblasNoTrans) {
    cuTransA = HIPBLAS_OP_N;
  }
  else if(TransA == CblasTrans){
    cuTransA = HIPBLAS_OP_T;
  }
  else {
    cuTransA = HIPBLAS_OP_C;
  }
  CUBLAS_CHECK(hipblasCgemv(Caffe::cublas_handle(), cuTransA, N, M, (const hipComplex*)&alpha,
      (const hipComplex*)A, N, (const hipComplex*)x, 1, (const hipComplex*)&beta, (hipComplex*)y, 1));
}

template <>
void caffe_gpu_gemv<std::complex<double> >(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const std::complex<double> alpha, const std::complex<double>* A, const std::complex<double>* x,
    const std::complex<double> beta, std::complex<double>* y) {
  hipblasOperation_t cuTransA;
  if(TransA == CblasNoTrans) {
    cuTransA = HIPBLAS_OP_N;
  }
  else if(TransA == CblasTrans){
    cuTransA = HIPBLAS_OP_T;
  }
  else {
    cuTransA = HIPBLAS_OP_C;
  }
  CUBLAS_CHECK(hipblasZgemv(Caffe::cublas_handle(), cuTransA, N, M, (const hipDoubleComplex*)&alpha,
      (const hipDoubleComplex*)A, N, (const hipDoubleComplex*)x, 1, (const hipDoubleComplex*)&beta, (hipDoubleComplex*)y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_axpy<std::complex<float> >(const int N, const std::complex<float> alpha, const std::complex<float>* X,
    std::complex<float>* Y) {
  CUBLAS_CHECK(hipblasCaxpy(Caffe::cublas_handle(), N, (const hipComplex*)&alpha, (const hipComplex*)X, 1, (hipComplex*)Y, 1));
}

template <>
void caffe_gpu_axpy<std::complex<double> >(const int N, const std::complex<double> alpha, const std::complex<double>* X,
    std::complex<double>* Y) {
  CUBLAS_CHECK(hipblasZaxpy(Caffe::cublas_handle(), N, (const hipDoubleComplex*)&alpha, (const hipDoubleComplex*)X, 1, (hipDoubleComplex*)Y, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<std::complex<float> >(const int N, const std::complex<float> alpha, std::complex<float> *X) {
  CUBLAS_CHECK(hipblasCscal(Caffe::cublas_handle(), N, (const hipComplex*)&alpha, (hipComplex*)X, 1));
}

template <>
void caffe_gpu_scal<std::complex<double> >(const int N, const std::complex<double> alpha, std::complex<double> *X) {
  CUBLAS_CHECK(hipblasZscal(Caffe::cublas_handle(), N, (const hipDoubleComplex*)&alpha, (hipDoubleComplex*)X, 1));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<std::complex<float> >(const int N, const std::complex<float> alpha, const std::complex<float>* X,
    const std::complex<float> beta, std::complex<float>* Y) {
  caffe_gpu_scal<std::complex<float> >(N, beta, Y);
  caffe_gpu_axpy<std::complex<float> >(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<std::complex<double> >(const int N, const std::complex<double> alpha, const std::complex<double>* X,
    const std::complex<double> beta, std::complex<double>* Y) {
  caffe_gpu_scal<std::complex<double> >(N, beta, Y);
  caffe_gpu_axpy<std::complex<double> >(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out, const bool conj_x) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out, const bool conj_x) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<std::complex<float> >(const int n, const std::complex<float>* x, const std::complex<float>* y,
    std::complex<float>* out, const bool conj_x) {
  if(conj_x) {
    CUBLAS_CHECK(hipblasCdotc(Caffe::cublas_handle(), n, (const hipComplex*)x, 1, (const hipComplex*)y, 1, (hipComplex*)out));
  } else {
    CUBLAS_CHECK(hipblasCdotu(Caffe::cublas_handle(), n, (const hipComplex*)x, 1, (const hipComplex*)y, 1, (hipComplex*)out));
  }
}

template <>
void caffe_gpu_dot<std::complex<double> >(const int n, const std::complex<double>* x, const std::complex<double>* y,
    std::complex<double>* out, const bool conj_x) {
  if(conj_x) {
    CUBLAS_CHECK(hipblasZdotc(Caffe::cublas_handle(), n, (const hipDoubleComplex*)x, 1, (const hipDoubleComplex*)y, 1, (hipDoubleComplex*)out));
  } else {
    CUBLAS_CHECK(hipblasZdotu(Caffe::cublas_handle(), n, (const hipDoubleComplex*)x, 1, (const hipDoubleComplex*)y, 1, (hipDoubleComplex*)out));
  }
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<std::complex<float> >(const int n, const std::complex<float>* x, std::complex<float>* y) {
  float result;
  CUBLAS_CHECK(hipblasScasum(Caffe::cublas_handle(), n, (hipComplex*)x, 1, &result));
  *y = std::complex<float>(result);
}

template <>
void caffe_gpu_asum<std::complex<double> >(const int n, const std::complex<double>* x, std::complex<double>* y) {
  double result;
  CUBLAS_CHECK(hipblasDzasum(Caffe::cublas_handle(), n, (hipDoubleComplex*)x, 1, &result));
  *y = std::complex<double>(result);
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<std::complex<float> >(const int n, const std::complex<float> alpha, const std::complex<float> *x,
                            std::complex<float>* y) {
  CUBLAS_CHECK(hipblasCcopy(Caffe::cublas_handle(), n, (const hipComplex*)x, 1, (hipComplex*)y, 1));
  CUBLAS_CHECK(hipblasCscal(Caffe::cublas_handle(), n, (const hipComplex*)&alpha, (hipComplex*)y, 1));
}

template <>
void caffe_gpu_scale<std::complex<double> >(const int n, const std::complex<double> alpha, const std::complex<double> *x,
                            std::complex<double>* y) {
  CUBLAS_CHECK(hipblasZcopy(Caffe::cublas_handle(), n, (const hipDoubleComplex*)x, 1, (hipDoubleComplex*)y, 1));
  CUBLAS_CHECK(hipblasZscal(Caffe::cublas_handle(), n, (const hipDoubleComplex*)&alpha, (hipDoubleComplex*)y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <>
void caffe_gpu_set(const int N, const std::complex<float> alpha, std::complex<float>* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, *(const hipComplex*)&alpha, (hipComplex*)Y);
}

template <>
void caffe_gpu_set(const int N, const std::complex<double> alpha, std::complex<double>* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, *(const hipDoubleComplex*)&alpha, (hipDoubleComplex*)Y);
}


template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template __global__ void add_scalar_kernel(const int n, const float alpha, float* y);
template __global__ void add_scalar_kernel(const int n, const double alpha, double* y);

template <>
__global__ void add_scalar_kernel(const int n, const hipComplex alpha, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCaddf(y[index],alpha);
  }
}

template <>
__global__ void add_scalar_kernel(const int n, const hipDoubleComplex alpha, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCadd(y[index],alpha);
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const std::complex<float> alpha, std::complex<float>* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, *(const hipComplex*)&alpha, (hipComplex*)Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const std::complex<double> alpha, std::complex<double>* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, *(const hipDoubleComplex*)&alpha, (hipDoubleComplex*)Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template __global__ void add_kernel(const int n, const float* a, const float* b, float* y);
template __global__ void add_kernel(const int n, const double* a, const double* b, double* y);

template <>
__global__ void add_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCaddf(a[index],b[index]);
  }
}

template <>
__global__ void add_kernel(const int n, const hipDoubleComplex* a, const hipDoubleComplex* b, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCadd(a[index],b[index]);
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<std::complex<float> >(const int N, const std::complex<float>* a, const std::complex<float>* b,
    std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (const hipComplex*)b, (hipComplex*)y);
}

template <>
void caffe_gpu_add<std::complex<double> >(const int N, const std::complex<double>* a, const std::complex<double>* b,
    std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (const hipDoubleComplex*)b, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template __global__ void sub_kernel(const int n, const float* a, const float* b, float* y);
template __global__ void sub_kernel(const int n, const double* a, const double* b, double* y);

template <>
__global__ void sub_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCsubf(a[index],b[index]);
  }
}

template <>
__global__ void sub_kernel(const int n, const hipDoubleComplex* a, const hipDoubleComplex* b, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCsub(a[index],b[index]);
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<std::complex<float> >(const int N, const std::complex<float>* a, const std::complex<float>* b,
    std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (const hipComplex*)b, (hipComplex*)y);
}

template <>
void caffe_gpu_sub<std::complex<double> >(const int N, const std::complex<double>* a, const std::complex<double>* b,
    std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (const hipDoubleComplex*)b, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template __global__ void mul_kernel(const int n, const float* a, const float* b, float* y);
template __global__ void mul_kernel(const int n, const double* a, const double* b, double* y);

template <>
__global__ void mul_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCmulf(a[index],b[index]);
  }
}

template <>
__global__ void mul_kernel(const int n, const hipDoubleComplex* a, const hipDoubleComplex* b, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCmul(a[index],b[index]);
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<std::complex<float> >(const int N, const std::complex<float>* a, const std::complex<float>* b,
    std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (const hipComplex*)b, (hipComplex*)y);
}

template <>
void caffe_gpu_mul<std::complex<double> >(const int N, const std::complex<double>* a, const std::complex<double>* b,
    std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (const hipDoubleComplex*)b, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template __global__ void div_kernel(const int n, const float* a, const float* b, float* y);
template __global__ void div_kernel(const int n, const double* a, const double* b, double* y);

template <>
__global__ void div_kernel(const int n, const hipComplex* a, const hipComplex* b, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCdivf(a[index],b[index]);
  }
}

template <>
__global__ void div_kernel(const int n, const hipDoubleComplex* a, const hipDoubleComplex* b, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCdiv(a[index],b[index]);
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<std::complex<float> >(const int N, const std::complex<float>* a, const std::complex<float>* b,
    std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (const hipComplex*)b, (hipComplex*)y);
}

template <>
void caffe_gpu_div<std::complex<double> >(const int N, const std::complex<double>* a, const std::complex<double>* b,
    std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (const hipDoubleComplex*)b, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template __global__ void abs_kernel(const int n, const float* a, float* y);
template __global__ void abs_kernel(const int n, const double* a, double* y);

template <>
__global__ void abs_kernel(const int n, const hipComplex* a, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = make_hipFloatComplex(hipCabsf(a[index]),0);
  }
}

template <>
__global__ void abs_kernel(const int n, const hipDoubleComplex* a, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = make_hipDoubleComplex(hipCabs(a[index]),0);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<std::complex<float> >(const int N, const std::complex<float>* a, std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (hipComplex*)y);
}

template <>
void caffe_gpu_abs<std::complex<double> >(const int N, const std::complex<double>* a, std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (hipDoubleComplex*)y);
}

__global__ void abs_kernel(const int n, const hipComplex* a, float* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCabsf(a[index]);
  }
}

__global__ void abs_kernel(const int n, const hipDoubleComplex* a, double* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipCabs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const std::complex<float>* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const std::complex<double>* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template __global__ void exp_kernel(const int n, const float* a, float* y);
template __global__ void exp_kernel(const int n, const double* a, double* y);

__device__ void caffe_gpu_complex_exp(const hipComplex a, hipComplex &b) {
    float exp_real = expf(a.x);
    float sin_imag;
    float cos_imag;
    sincosf(a.y, &sin_imag, &cos_imag);
    b.x = exp_real*cos_imag;
    b.y = exp_real*sin_imag;
}

__device__ void caffe_gpu_complex_exp(const hipDoubleComplex a, hipDoubleComplex &b) {
    double exp_real = exp(a.x);
    double sin_imag;
    double cos_imag;
    sincos(a.y, &sin_imag, &cos_imag);
    b.x = exp_real*cos_imag;
    b.y = exp_real*sin_imag;
}

__global__ void exp_kernel(const int n, const hipComplex* a, hipComplex* b) {
  CUDA_KERNEL_LOOP(index, n) {
    caffe_gpu_complex_exp(a[index],b[index]);
  }
}

__global__ void exp_kernel(const int n, const hipDoubleComplex* a, hipDoubleComplex* b) {
  CUDA_KERNEL_LOOP(index, n) {
	caffe_gpu_complex_exp(a[index],b[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<std::complex<float> >(const int N, const std::complex<float>* a, std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (hipComplex*)y);
}

template <>
void caffe_gpu_exp<std::complex<double> >(const int N, const std::complex<double>* a, std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template __global__ void log_kernel(const int n, const float* a, float* y);
template __global__ void log_kernel(const int n, const double* a, double* y);

template <>
__global__ void log_kernel(const int n, const hipComplex* a, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index].x = hipCabsf(a[index]);
    y[index].y = atan2f(a[index].y, a[index].x);
  }
}

template <>
__global__ void log_kernel(const int n, const hipDoubleComplex* a, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index].x = hipCabs(a[index]);
    y[index].y = atan2(a[index].y, a[index].x);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<std::complex<float> >(const int N, const std::complex<float>* a, std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<hipComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (hipComplex*)y);
}

template <>
void caffe_gpu_log<std::complex<double> >(const int N, const std::complex<double>* a, std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<hipDoubleComplex><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (hipDoubleComplex*)y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

__global__ void powx_kernel(const int n, const hipComplex* a,
    const float alpha, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
	// y = a^alpha
	//   = exp(alpha*log(a))
	// alpha*log(a) = {alpha*abs(a), alpha*arg(z)}
	// y = exp(alpha*log(a))
	//   = {exp(alpha*abs(a)*cos(alpha*arg(z)), exp(alpha*abs(a)*sign(alpha*arg(z))}
	float exp_alpha_abs_a = expf(alpha * hipCabsf(a[index]));
	float theta = atan2f(a[index].y, a[index].x);
	float cos_alpha_theta;
	float sin_alpha_theta;
	sincosf(alpha*theta, &sin_alpha_theta, &cos_alpha_theta);
    y[index].x = exp_alpha_abs_a * cos_alpha_theta;
    y[index].y = exp_alpha_abs_a * sin_alpha_theta;
  }
}

__global__ void powx_kernel(const int n, const hipDoubleComplex* a,
    const double alpha, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
	// y = a^alpha
	//   = exp(alpha*log(a))
	// alpha*log(a) = {alpha*abs(a), alpha*arg(z)}
	// y = exp(alpha*log(a))
	//   = {exp(alpha*abs(a)*cos(alpha*arg(z)), exp(alpha*abs(a)*sign(alpha*arg(z))}
	double exp_alpha_abs_a = expf(alpha * hipCabs(a[index]));
	double theta = atan2(a[index].y, a[index].x);
	double cos_alpha_theta;
	double sin_alpha_theta;
	sincos(alpha*theta, &sin_alpha_theta, &cos_alpha_theta);
    y[index].x = exp_alpha_abs_a * cos_alpha_theta;
    y[index].y = exp_alpha_abs_a * sin_alpha_theta;
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const std::complex<float>* a,
    const float alpha, std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, alpha, (hipComplex*)y);
}

template <>
void caffe_gpu_powx<double>(const int N, const std::complex<double>* a,
    const double alpha, std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, alpha, (hipDoubleComplex*)y);
}

__global__ void conj_kernel(const int n, const hipComplex* a, hipComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipConjf(a[index]);
  }
}

__global__ void conj_kernel(const int n, const hipDoubleComplex* a, hipDoubleComplex* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hipConj(a[index]);
  }
}

template <>
void caffe_gpu_conj<std::complex<float> >(const int N, const std::complex<float>* a, std::complex<float>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  conj_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipComplex*)a, (hipComplex*)y);
}

template <>
void caffe_gpu_conj<std::complex<double> >(const int N, const std::complex<double>* a, std::complex<double>* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  conj_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, (const hipDoubleComplex*)a, (hipDoubleComplex*)y);
}


DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<std::complex<float> >(const int n, const std::complex<float> a,
		const std::complex<float> b, std::complex<float>* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), reinterpret_cast<float*>(r), n*2));
  const std::complex<float> range = b - a;
  // Zero out imaginary
  for(int i = 0; i < n; i++) {
	  r[i] = std::complex<float>(std::real(r[i]),0);
  }
  if (std::real(range) != 1 && std::imag(range) != 0) {
    caffe_gpu_scal(n, range, r);
  }
  if (std::real(a) != 0 && std::imag(a) != 0) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<std::complex<double> >(const int n, const std::complex<double> a,
		const std::complex<double> b, std::complex<double>* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), reinterpret_cast<double*>(r), n*2));
  const std::complex<double> range = b - a;
  // Zero out imaginary
  for(int i = 0; i < n; i++) {
	  r[i] = std::complex<double>(std::real(r[i]),0);
  }
  if (std::real(range) != 1 && std::imag(range) != 0) {
    caffe_gpu_scal(n, range, r);
  }
  if (std::real(a) != 0 && std::imag(a) != 0) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const std::complex<float> mu, const std::complex<float> sigma,
                            std::complex<float>* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), reinterpret_cast<float*>(r), n*2,
    		  std::real(mu), std::real(sigma)));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const std::complex<double> mu, const std::complex<double> sigma,
                            std::complex<double>* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), reinterpret_cast<double*>(r), n*2,
    		  std::real(mu), std::real(sigma)));
}


}  // namespace caffe
